
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/cast.h"
#include "dragon/utils/math/blas.h"

namespace dragon {

namespace math {

namespace {

template <typename T>
__global__ void _Scale(const int n, const T alpha, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = x[i] * alpha;
  }
}

template <typename T>
__global__ void _Axpy(const int n, const T alpha, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] += (alpha * x[i]);
  }
}

template <typename T>
__global__ void
_Axpby(const int n, const T alpha, const T* x, const T beta, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = (alpha * x[i] + beta * y[i]);
  }
}

template <>
__global__ void _Axpby<half>(
    const int n,
    const half alpha,
    const half* x,
    const half beta,
    half* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
    y[i] = __hadd(__hmul(alpha, x[i]), __hmul(beta, y[i]));
#endif
  }
}

template <>
__global__ void _Axpby<half2>(
    const int n,
    const half2 alpha,
    const half2* x,
    const half2 beta,
    half2* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
    y[i] = __hadd2(__hmul2(alpha, x[i]), __hmul2(beta, y[i]));
#endif
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_SCALE_FUNC(T)                                                \
  template <>                                                               \
  DRAGON_API void Scale<T, CUDAContext>(                                    \
      const int n, const float alpha, const T* x, T* y, CUDAContext* ctx) { \
    T _alpha_ = (T)alpha;                                                   \
    if (_alpha_ == T(1)) {                                                  \
      if (x != y) {                                                         \
        cudaMemcpyAsync(                                                    \
            y,                                                              \
            x,                                                              \
            sizeof(T) * n,                                                  \
            cudaMemcpyDeviceToDevice,                                       \
            ctx->cuda_stream());                                            \
      }                                                                     \
      return;                                                               \
    }                                                                       \
    _Scale<<<CUDA_BLOCKS(n), CUDA_THREADS, 0, ctx->cuda_stream()>>>(        \
        n, _alpha_, x, y);                                                  \
  }

DEFINE_SCALE_FUNC(int8_t);
DEFINE_SCALE_FUNC(uint8_t);
DEFINE_SCALE_FUNC(int);
DEFINE_SCALE_FUNC(int64_t);
#undef DEFINE_SCALE_FUNC

#define DEFINE_SCALE_FUNC(T, cublas_func)                                      \
  template <>                                                                  \
  DRAGON_API void Scale<T, CUDAContext>(                                       \
      const int n, const float alpha, const T* x, T* y, CUDAContext* ctx) {    \
    if (x != y) {                                                              \
      CUDA_CHECK(cudaMemcpyAsync(                                              \
          y, x, sizeof(T) * n, cudaMemcpyDeviceToDevice, ctx->cuda_stream())); \
    }                                                                          \
    if (alpha != 1.f) {                                                        \
      T scale = (T)alpha;                                                      \
      CUBLAS_CHECK(cublasSetPointerMode(                                       \
          ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST));                    \
      CUBLAS_CHECK(cublas_func(ctx->cublas_handle(), n, &scale, y, 1));        \
    }                                                                          \
  }

template <>
DRAGON_API void Scale<float16, CUDAContext>(
    const int n,
    const float alpha,
    const float16* x,
    float16* y,
    CUDAContext* ctx) {
  if (x != y) {
    CUDA_CHECK(cudaMemcpyAsync(
        y,
        x,
        sizeof(float16) * n,
        cudaMemcpyDeviceToDevice,
        ctx->cuda_stream()));
  }
  if (alpha != 1.f) {
    CUBLAS_CHECK(
        cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST));
    CUBLAS_CHECK(cublasScalEx(
        ctx->cublas_handle(),
        n,
        &alpha,
        CUDA_R_32F,
        y,
        CUDA_R_16F,
        1,
        CUDA_R_32F));
  }
}

DEFINE_SCALE_FUNC(float, cublasSscal);
DEFINE_SCALE_FUNC(double, cublasDscal);
#undef DEFINE_SCALE_FUNC

#define DEFINE_COPY_FUNC(T)                                                    \
  template <>                                                                  \
  DRAGON_API void Copy<T, CUDAContext>(                                        \
      const int n, const T* x, T* y, CUDAContext* ctx) {                       \
    if (x != y && n > 0) {                                                     \
      CUDA_CHECK(cudaMemcpyAsync(                                              \
          y, x, n * sizeof(T), cudaMemcpyDeviceToDevice, ctx->cuda_stream())); \
    }                                                                          \
  }

DEFINE_COPY_FUNC(bool);
DEFINE_COPY_FUNC(int8_t);
DEFINE_COPY_FUNC(uint8_t);
DEFINE_COPY_FUNC(int);
DEFINE_COPY_FUNC(int64_t);
DEFINE_COPY_FUNC(float16);
DEFINE_COPY_FUNC(float);
DEFINE_COPY_FUNC(double);
#undef DEFINE_COPY_FUNC

#define DEFINE_AXPY_FUNC(T)                                                 \
  template <>                                                               \
  DRAGON_API void Axpy<T, CUDAContext>(                                     \
      const int n, const float alpha, const T* x, T* y, CUDAContext* ctx) { \
    _Axpy<<<CUDA_BLOCKS(n), CUDA_THREADS, 0, ctx->cuda_stream()>>>(         \
        n, (T)alpha, x, y);                                                 \
  }

DEFINE_AXPY_FUNC(int8_t);
DEFINE_AXPY_FUNC(uint8_t);
DEFINE_AXPY_FUNC(int);
DEFINE_AXPY_FUNC(int64_t);
#undef DEFINE_AXPY_FUNC

#define DEFINE_AXPY_FUNC(T, cublas_func)                                       \
  template <>                                                                  \
  DRAGON_API void Axpy<T, CUDAContext>(                                        \
      const int n, const float alpha, const T* x, T* y, CUDAContext* ctx) {    \
    T scale = (T)alpha;                                                        \
    CUBLAS_CHECK(                                                              \
        cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST)); \
    CUBLAS_CHECK(cublas_func(ctx->cublas_handle(), n, &scale, x, 1, y, 1));    \
  }

template <>
DRAGON_API void Axpy<float16, CUDAContext>(
    const int n,
    const float alpha,
    const float16* x,
    float16* y,
    CUDAContext* ctx) {
  CUBLAS_CHECK(
      cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST));
  CUBLAS_CHECK(cublasAxpyEx(
      ctx->cublas_handle(),
      n,
      &alpha,
      CUDA_R_32F,
      x,
      CUDA_R_16F,
      1,
      y,
      CUDA_R_16F,
      1,
      CUDA_R_32F));
}

DEFINE_AXPY_FUNC(float, cublasSaxpy);
DEFINE_AXPY_FUNC(double, cublasDaxpy);
#undef DEFINE_AXPY_FUNC

#define DEFINE_AXPBY_FUNC(T)                                         \
  template <>                                                        \
  DRAGON_API void Axpby<T, CUDAContext>(                             \
      const int n,                                                   \
      const float alpha,                                             \
      const T* x,                                                    \
      const float beta,                                              \
      T* y,                                                          \
      CUDAContext* ctx) {                                            \
    _Axpby<<<CUDA_BLOCKS(n), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        n, (T)alpha, x, (T)beta, y);                                 \
  }

template <>
DRAGON_API void Axpby<float16, CUDAContext>(
    const int n,
    const float alpha,
    const float16* x,
    const float beta,
    float16* y,
    CUDAContext* ctx) {
  if ((n & 1) == 0) {
    _Axpby<<<CUDA_BLOCKS(n >> 1), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
        n >> 1,
        cast::to<half2>(alpha),
        reinterpret_cast<const half2*>(x),
        cast::to<half2>(beta),
        reinterpret_cast<half2*>(y));
  } else {
    _Axpby<<<CUDA_BLOCKS(n), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
        n,
        cast::to<half>(alpha),
        reinterpret_cast<const half*>(x),
        cast::to<half>(beta),
        reinterpret_cast<half*>(y));
  }
}

DEFINE_AXPBY_FUNC(int8_t);
DEFINE_AXPBY_FUNC(uint8_t);
DEFINE_AXPBY_FUNC(int);
DEFINE_AXPBY_FUNC(int64_t);
DEFINE_AXPBY_FUNC(float);
DEFINE_AXPBY_FUNC(double);
#undef DEFINE_AXPBY_FUNC

#define DEFINE_DOT_FUNC(T, cublas_func)                                        \
  template <>                                                                  \
  DRAGON_API void Dot<T, CUDAContext>(                                         \
      const int n, const T* a, const T* b, T* y, CUDAContext* ctx) {           \
    CUBLAS_CHECK(cublasSetPointerMode(                                         \
        ctx->cublas_handle(), CUBLAS_POINTER_MODE_DEVICE));                    \
    CUBLAS_CHECK(cublas_func(ctx->cublas_handle(), n, a, 1, b, 1, y));         \
  }                                                                            \
  template <>                                                                  \
  DRAGON_API T Dot<T, CUDAContext>(                                            \
      const int n, const T* a, const T* b, CUDAContext* ctx) {                 \
    T y_host;                                                                  \
    CUBLAS_CHECK(                                                              \
        cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST)); \
    CUBLAS_CHECK(cublas_func(ctx->cublas_handle(), n, a, 1, b, 1, &y_host));   \
    return y_host;                                                             \
  }

template <>
DRAGON_API void Dot<float16, CUDAContext>(
    const int n,
    const float16* a,
    const float16* b,
    float16* y,
    CUDAContext* ctx) {
  CUBLAS_CHECK(
      cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_DEVICE));
  CUBLAS_CHECK(cublasDotEx(
      ctx->cublas_handle(),
      n,
      a,
      CUDA_R_16F,
      1,
      b,
      CUDA_R_16F,
      1,
      y,
      CUDA_R_16F,
      CUDA_R_32F));
}

DEFINE_DOT_FUNC(float, cublasSdot);
DEFINE_DOT_FUNC(double, cublasDdot);
#undef DEFINE_DOT_FUNC

#define DEFINE_ASUM_FUNC(T, cublas_func)                                       \
  template <>                                                                  \
  DRAGON_API void ASum<T, CUDAContext>(                                        \
      const int n, const T* x, T* y, CUDAContext* ctx) {                       \
    CUBLAS_CHECK(cublasSetPointerMode(                                         \
        ctx->cublas_handle(), CUBLAS_POINTER_MODE_DEVICE));                    \
    cublas_func(ctx->cublas_handle(), n, x, 1, y);                             \
  }                                                                            \
  template <>                                                                  \
  DRAGON_API T ASum<T, CUDAContext>(                                           \
      const int n, const T* x, CUDAContext* ctx) {                             \
    T y_host;                                                                  \
    CUBLAS_CHECK(                                                              \
        cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST)); \
    cublas_func(ctx->cublas_handle(), n, x, 1, &y_host);                       \
    return y_host;                                                             \
  }

DEFINE_ASUM_FUNC(float, cublasSasum);
DEFINE_ASUM_FUNC(double, cublasDasum);
#undef DEFINE_ASUM_FUNC

template <>
DRAGON_API void Gemv<float16, CUDAContext>(
    const CBLAS_TRANSPOSE TransA,
    const int M,
    const int N,
    const float alpha,
    const float16* A,
    const float16* x,
    const float beta,
    float16* y,
    CUDAContext* ctx,
    const string math_type) {
  auto cuTransA = TransA == CblasNoTrans ? CUBLAS_OP_T : CUBLAS_OP_N;
  int m = cuTransA == CUBLAS_OP_N ? N : M;
  int k = cuTransA == CUBLAS_OP_N ? M : N;
  int LDA = cuTransA == CUBLAS_OP_N ? m : k;
  int LDC = m;
  CUBLAS_CHECK(
      cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST));
  if (math_type == "float32") {
#if CUDA_VERSION >= 9000
    if (TENSOR_CORE_AVAILABLE()) {
      // GEMV + MATH32 + TENSOR-CORE
      CUBLAS_CHECK(cublasGemmEx(
          ctx->cublas_handle(),
          cuTransA,
          CUBLAS_OP_N,
          m,
          1,
          k,
          &alpha,
          A,
          CUDA_R_16F,
          LDA,
          x,
          CUDA_R_16F,
          k,
          &beta,
          y,
          CUDA_R_16F,
          LDC,
          CUDA_R_32F,
          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    } else {
      // GEMV + MATH32 + DEFAULT
      CUBLAS_CHECK(cublasSgemmEx(
          ctx->cublas_handle(),
          cuTransA,
          CUBLAS_OP_N,
          m,
          1,
          k,
          &alpha,
          A,
          CUDA_R_16F,
          LDA,
          x,
          CUDA_R_16F,
          k,
          &beta,
          y,
          CUDA_R_16F,
          LDC));
    }
#else
    CUBLAS_CHECK(cublasSgemmEx(
        ctx->cublas_handle(),
        cuTransA,
        CUBLAS_OP_N,
        m,
        1,
        k,
        &alpha,
        A,
        CUDA_R_16F,
        LDA,
        x,
        CUDA_R_16F,
        k,
        &beta,
        y,
        CUDA_R_16F,
        LDC));
#endif
  } else if (math_type == "float16") {
    const half alpha_half = cast::to<half>(alpha);
    const half beta_half = cast::to<half>(beta);
#if CUDA_VERSION >= 9000
    if (TENSOR_CORE_AVAILABLE()) {
      // GEMV + MATH16 + TENSOR-CORE
      CUBLAS_CHECK(cublasGemmEx(
          ctx->cublas_handle(),
          cuTransA,
          CUBLAS_OP_N,
          m,
          1,
          k,
          &alpha_half,
          A,
          CUDA_R_16F,
          LDA,
          x,
          CUDA_R_16F,
          k,
          &beta_half,
          y,
          CUDA_R_16F,
          LDC,
          CUDA_R_16F,
          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    } else {
      // GEMV + MATH16 + DEFAULT
      CUBLAS_CHECK(cublasHgemm(
          ctx->cublas_handle(),
          cuTransA,
          CUBLAS_OP_N,
          m,
          1,
          k,
          &alpha_half,
          reinterpret_cast<const half*>(A),
          LDA,
          reinterpret_cast<const half*>(x),
          k,
          &beta_half,
          reinterpret_cast<half*>(y),
          LDC));
    }
#else
    CUBLAS_CHECK(cublasHgemm(
        ctx->cublas_handle(),
        cuTransA,
        CUBLAS_OP_N,
        m,
        1,
        k,
        &alpha_half,
        reinterpret_cast<const half*>(A),
        LDA,
        reinterpret_cast<const half*>(x),
        k,
        &beta_half,
        reinterpret_cast<half*>(y),
        LDC));
#endif
  } else {
    LOG(FATAL) << "Unknown math type: " << math_type;
  }
}

template <>
DRAGON_API void Gemv<float, CUDAContext>(
    const CBLAS_TRANSPOSE TransA,
    const int M,
    const int N,
    const float alpha,
    const float* A,
    const float* x,
    const float beta,
    float* y,
    CUDAContext* ctx,
    const string math_type) {
  auto cuTransA = TransA == CblasNoTrans ? CUBLAS_OP_T : CUBLAS_OP_N;
  CUBLAS_CHECK(
      cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST));
  CUBLAS_CHECK(cublasSgemv(
      ctx->cublas_handle(), cuTransA, N, M, &alpha, A, N, x, 1, &beta, y, 1));
}

template <>
DRAGON_API void Gemv<double, CUDAContext>(
    const CBLAS_TRANSPOSE TransA,
    const int M,
    const int N,
    const float alpha,
    const double* A,
    const double* x,
    const float beta,
    double* y,
    CUDAContext* ctx,
    const string math_type) {
  auto cuTransA = TransA == CblasNoTrans ? CUBLAS_OP_T : CUBLAS_OP_N;
  const auto alpha64 = static_cast<double>(alpha);
  const auto beta64 = static_cast<double>(beta);
  CUBLAS_CHECK(
      cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST));
  CUBLAS_CHECK(cublasDgemv(
      ctx->cublas_handle(),
      cuTransA,
      N,
      M,
      &alpha64,
      A,
      N,
      x,
      1,
      &beta64,
      y,
      1));
}

template <>
DRAGON_API void Gemm<float16, CUDAContext>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float16* A,
    const float16* B,
    const float beta,
    float16* C,
    CUDAContext* ctx,
    const std::string math_type) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  auto cuTransA = TransA == CblasNoTrans ? CUBLAS_OP_N : CUBLAS_OP_T;
  auto cuTransB = TransB == CblasNoTrans ? CUBLAS_OP_N : CUBLAS_OP_T;
  CUBLAS_CHECK(
      cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST));
  if (math_type == "float32") {
#if CUDA_VERSION >= 9000
    if (TENSOR_CORE_AVAILABLE()) {
      // GEMM + MATH32 + TENSOR-CORE
      CUBLAS_CHECK(cublasGemmEx(
          ctx->cublas_handle(),
          cuTransB,
          cuTransA,
          N,
          M,
          K,
          &alpha,
          B,
          CUDA_R_16F,
          ldb,
          A,
          CUDA_R_16F,
          lda,
          &beta,
          C,
          CUDA_R_16F,
          N,
          CUDA_R_32F,
          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    } else {
      // GEMM + MATH32 + DEFAULT
      CUBLAS_CHECK(cublasSgemmEx(
          ctx->cublas_handle(),
          cuTransB,
          cuTransA,
          N,
          M,
          K,
          &alpha,
          B,
          CUDA_R_16F,
          ldb,
          A,
          CUDA_R_16F,
          lda,
          &beta,
          C,
          CUDA_R_16F,
          N));
    }
#else
    CUBLAS_CHECK(cublasSgemmEx(
        ctx->cublas_handle(),
        cuTransB,
        cuTransA,
        N,
        M,
        K,
        &alpha,
        B,
        CUDA_R_16F,
        ldb,
        A,
        CUDA_R_16F,
        lda,
        &beta,
        C,
        CUDA_R_16F,
        N));
#endif
  } else if (math_type == "float16") {
    const half alpha_half = cast::to<half>(alpha);
    const half beta_half = cast::to<half>(beta);
#if CUDA_VERSION >= 9000
    if (TENSOR_CORE_AVAILABLE()) {
      // GEMM + MATH16 + TENSOR-CORE
      CUBLAS_CHECK(cublasGemmEx(
          ctx->cublas_handle(),
          cuTransB,
          cuTransA,
          N,
          M,
          K,
          &alpha_half,
          B,
          CUDA_R_16F,
          ldb,
          A,
          CUDA_R_16F,
          lda,
          &beta_half,
          C,
          CUDA_R_16F,
          N,
          CUDA_R_16F,
          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    } else {
      // GEMM + MATH16 + DEFAULT
      CUBLAS_CHECK(cublasHgemm(
          ctx->cublas_handle(),
          cuTransB,
          cuTransA,
          N,
          M,
          K,
          &alpha_half,
          reinterpret_cast<const half*>(B),
          ldb,
          reinterpret_cast<const half*>(A),
          lda,
          &beta_half,
          reinterpret_cast<half*>(C),
          N));
    }
#else
    CUBLAS_CHECK(cublasHgemm(
        ctx->cublas_handle(),
        cuTransB,
        cuTransA,
        N,
        M,
        K,
        &alpha_half,
        reinterpret_cast<const half*>(B),
        ldb,
        reinterpret_cast<const half*>(A),
        lda,
        &beta_half,
        reinterpret_cast<half*>(C),
        N));
#endif
  } else {
    LOG(FATAL) << "Unknown math type: " << math_type;
  }
}

template <>
DRAGON_API void Gemm<float, CUDAContext>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const float* B,
    const float beta,
    float* C,
    CUDAContext* ctx,
    const string math_type) {
  int lda = TransA == CblasNoTrans ? K : M;
  int ldb = TransB == CblasNoTrans ? N : K;
  auto cuTransA = TransA == CblasNoTrans ? CUBLAS_OP_N : CUBLAS_OP_T;
  auto cuTransB = TransB == CblasNoTrans ? CUBLAS_OP_N : CUBLAS_OP_T;
  CUBLAS_CHECK(
      cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST));
  CUBLAS_CHECK(cublasSgemm(
      ctx->cublas_handle(),
      cuTransB,
      cuTransA,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      A,
      lda,
      &beta,
      C,
      N));
}

template <>
DRAGON_API void Gemm<double, CUDAContext>(
    const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const double* A,
    const double* B,
    const float beta,
    double* C,
    CUDAContext* ctx,
    const string math_type) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  auto cuTransA = TransA == CblasNoTrans ? CUBLAS_OP_N : CUBLAS_OP_T;
  auto cuTransB = TransB == CblasNoTrans ? CUBLAS_OP_N : CUBLAS_OP_T;
  const auto alpha64 = static_cast<double>(alpha);
  const auto beta64 = static_cast<double>(beta);
  CUBLAS_CHECK(
      cublasSetPointerMode(ctx->cublas_handle(), CUBLAS_POINTER_MODE_HOST));
  CUBLAS_CHECK(cublasDgemm(
      ctx->cublas_handle(),
      cuTransB,
      cuTransA,
      N,
      M,
      K,
      &alpha64,
      B,
      ldb,
      A,
      lda,
      &beta64,
      C,
      N));
}

} // namespace math

} // namespace dragon

#endif // USE_CUDA
