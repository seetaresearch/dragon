
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, int D>
__global__ void _Transpose(
    const int N,
    const int num_dims,
    const SimpleArray<int, D> X_strides,
    const SimpleArray<int, D> Y_dims,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, N) {
    int xi = 0, tmp = yi;
    for (int d = num_dims - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(Y_dims.data[d], tmp, &tmp, &r);
      xi += r * X_strides.data[d];
    }
    y[yi] = x[xi];
  }
}

template <typename T, int D>
__global__ void _TransposeGrad(
    const int N,
    const int num_dims,
    const SimpleArray<int, D> X_strides,
    const SimpleArray<int, D> Y_dims,
    const T* dy,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(yi, N) {
    int xi = 0, tmp = yi;
    for (int d = num_dims - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(Y_dims.data[d], tmp, &tmp, &r);
      xi += r * X_strides.data[d];
    }
    dx[xi] = dy[yi];
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(name, T)                               \
  template <>                                                         \
  void name<T, CUDAContext>(                                          \
      const int num_dims,                                             \
      const int64_t* x_strides,                                       \
      const int64_t* y_dims,                                          \
      const T* x,                                                     \
      T* y,                                                           \
      CUDAContext* ctx) {                                             \
    CUDA_TENSOR_DIMS_CHECK(num_dims);                                 \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> X_strides, Y_dims;         \
    const auto N = std::accumulate(                                   \
        y_dims, y_dims + num_dims, 1, std::multiplies<int64_t>());    \
    for (int i = 0; i < num_dims; ++i) {                              \
      X_strides.data[i] = x_strides[i];                               \
      Y_dims.data[i] = y_dims[i];                                     \
    }                                                                 \
    _##name<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        N, num_dims, X_strides, Y_dims, x, y);                        \
  }

DEFINE_KERNEL_LAUNCHER(Transpose, bool);
DEFINE_KERNEL_LAUNCHER(Transpose, uint8_t);
DEFINE_KERNEL_LAUNCHER(Transpose, int8_t);
DEFINE_KERNEL_LAUNCHER(Transpose, int);
DEFINE_KERNEL_LAUNCHER(Transpose, int64_t);
DEFINE_KERNEL_LAUNCHER(Transpose, float16);
DEFINE_KERNEL_LAUNCHER(Transpose, float);
DEFINE_KERNEL_LAUNCHER(Transpose, double);
DEFINE_KERNEL_LAUNCHER(TransposeGrad, float16);
DEFINE_KERNEL_LAUNCHER(TransposeGrad, float);
DEFINE_KERNEL_LAUNCHER(TransposeGrad, double);
#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon

#endif // USE_CUDA
