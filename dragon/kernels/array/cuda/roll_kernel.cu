#include "hip/hip_runtime.h"
#include "dragon/kernels/array/op_kernels.h"
#include "dragon/utils/math_functions.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, int D>
__global__ void _Roll(
    const int N,
    const SimpleArray<int, D> X_shifts,
    const SimpleArray<int, D> X_strides,
    const SimpleArray<int, D> Y_dims,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, N) {
    int xi = 0, tmp = yi;
#pragma unroll
    for (int d = D - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(Y_dims.data[d], tmp, &tmp, &r);
      r -= X_shifts.data[d];
      r = (r < 0 ? r + Y_dims.data[d] : r) % Y_dims.data[d];
      xi += r * X_strides.data[d];
    }
    y[yi] = x[xi];
  }
}

template <typename T, int D>
void _RollImpl(
    const int64_t* x_shifts,
    const int64_t* x_strides,
    const int64_t* y_dims,
    const T* x,
    T* y,
    HIPContext* ctx) {
  SimpleArray<int, D> X_shifts, X_strides, Y_dims;
  const auto N = math::utils::Prod(D, y_dims);
  for (int i = 0; i < D; ++i) {
    X_shifts.data[i] = x_shifts[i];
    X_strides.data[i] = x_strides[i];
    Y_dims.data[i] = y_dims[i];
  }
  _Roll<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(
      N, X_shifts, X_strides, Y_dims, x, y);
}

} // namespace

#define DEFINE_KERNEL_LAUNCHER(T)                                        \
  template <>                                                            \
  void Roll<T, HIPContext>(                                             \
      const int num_dims,                                                \
      const int64_t* x_shifts,                                           \
      const int64_t* x_strides,                                          \
      const int64_t* y_dims,                                             \
      const T* x,                                                        \
      T* y,                                                              \
      HIPContext* ctx) {                                                \
    CUDA_TENSOR_DIMS_CHECK(num_dims);                                    \
    DISPATCH_FUNC_BY_VALUE_WITH_TYPE_1(                                  \
        _RollImpl, T, num_dims, x_shifts, x_strides, y_dims, x, y, ctx); \
  }

DEFINE_KERNEL_LAUNCHER(bool);
DEFINE_KERNEL_LAUNCHER(uint8_t);
DEFINE_KERNEL_LAUNCHER(int8_t);
DEFINE_KERNEL_LAUNCHER(int);
DEFINE_KERNEL_LAUNCHER(int64_t);
DEFINE_KERNEL_LAUNCHER(float16);
DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);
#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon
