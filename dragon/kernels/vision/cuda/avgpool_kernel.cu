#include "hip/hip_runtime.h"
#include "dragon/kernels/vision/op_kernels.h"
#include "dragon/utils/math_functions.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, typename AccT>
__global__ void _AvgPool2dNCHW(
    const int nthreads,
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_h,
    const int pad_w,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int wout = yi % out_w;
    const int hout = yi / out_w % out_h;
    const int c = yi / out_w / out_h % C;
    const int n = yi / out_w / out_h / C;
    const int hstart = hout * stride_h - pad_h;
    const int wstart = wout * stride_w - pad_w;
    int hend = min(hstart + kernel_h, H + pad_h);
    int wend = min(wstart + kernel_w, W + pad_w);
    const AccT area = (hend - hstart) * (wend - wstart);
    hend = min(hend, H), wend = min(wend, W);
    const T* offset_x = x + (n * C + c) * H * W;
    AccT val = AccT(0);
    for (int h = max(hstart, 0); h < hend; ++h) {
      for (int w = max(wstart, 0); w < wend; ++w) {
        val += math::utils::LDGC<AccT>(offset_x + h * W + w);
      }
    }
    y[yi] = val / area;
  }
}

template <typename T, typename AccT>
__global__ void _AvgPool2dNHWC(
    const int nthreads,
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_h,
    const int pad_w,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int c = yi % C;
    const int wout = yi / C % out_w;
    const int hout = yi / C / out_w % out_h;
    const int n = yi / C / out_w / out_h;
    const int hstart = hout * stride_h - pad_h;
    const int wstart = wout * stride_w - pad_w;
    int hend = min(hstart + kernel_h, H + pad_h);
    int wend = min(wstart + kernel_w, W + pad_w);
    const AccT area = (hend - hstart) * (wend - wstart);
    hend = min(hend, H), wend = min(wend, W);
    const T* offset_x = x + n * H * W * C + c;
    AccT val = AccT(0);
    for (int h = max(hstart, 0); h < hend; ++h) {
      for (int w = max(wstart, 0); w < wend; ++w) {
        val += math::utils::LDGC<AccT>(offset_x + (h * W + w) * C);
      }
    }
    y[yi] = val / area;
  }
}

template <typename T, typename AccT>
__global__ void _AvgPool2dGradNCHW(
    const int nthreads,
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_h,
    const int pad_w,
    const T* dy,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(xi, nthreads) {
    const int w = xi % W + pad_w;
    const int h = xi / W % H + pad_h;
    const int c = xi / W / H % C;
    const int n = xi / W / H / C;
    const int out_hstart = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
    const int out_wstart = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
    const int out_hend = min(h / stride_h + 1, out_h);
    const int out_wend = min(w / stride_w + 1, out_w);
    const T* offset_dy = dy + (n * C + c) * out_h * out_w;
    AccT val = AccT(0);
    for (int hout = out_hstart; hout < out_hend; ++hout) {
      const int hstart = hout * stride_h - pad_h;
      const int hend = min(hstart + kernel_h, H + pad_h);
      for (int wout = out_wstart; wout < out_wend; ++wout) {
        const int wstart = wout * stride_w - pad_w;
        const int wend = min(wstart + kernel_w, W + pad_w);
        const AccT area = (hend - hstart) * (wend - wstart);
        val += math::utils::LDGC<AccT>(offset_dy + hout * out_w + wout) / area;
      }
    }
    dx[xi] = val;
  }
}

template <typename T, typename AccT>
__global__ void _AvgPool2dGradNHWC(
    const int nthreads,
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_h,
    const int pad_w,
    const T* dy,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(xi, nthreads) {
    const int c = xi % C;
    const int w = xi / C % W + pad_w;
    const int h = xi / C / W % H + pad_h;
    const int n = xi / C / W / H;
    const int out_hstart = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
    const int out_wstart = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
    const int out_hend = min(h / stride_h + 1, out_h);
    const int out_wend = min(w / stride_w + 1, out_w);
    const T* offset_dy = dy + n * out_h * out_w * C + c;
    AccT val = AccT(0);
    for (int hout = out_hstart; hout < out_hend; ++hout) {
      const int hstart = hout * stride_h - pad_h;
      const int hend = min(hstart + kernel_h, H + pad_h);
      for (int wout = out_wstart; wout < out_wend; ++wout) {
        const int wstart = wout * stride_w - pad_w;
        const int wend = min(wstart + kernel_w, W + pad_w);
        const AccT area = (hend - hstart) * (wend - wstart); // clang-format off
        val += math::utils::LDGC<AccT>(offset_dy + (hout * out_w + wout) * C) / area;
      } // clang-format on
    }
    dx[xi] = val;
  }
}

template <typename T, typename AccT>
__global__ void _AvgPool3dNCHW(
    const int nthreads,
    const int C,
    const int D,
    const int H,
    const int W,
    const int out_d,
    const int out_h,
    const int out_w,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_d,
    const int pad_h,
    const int pad_w,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    int tmp = yi / out_w;
    const int wout = yi % out_w;
    const int hout = tmp % out_h;
    tmp /= out_h;
    const int dout = tmp % out_d;
    tmp /= out_d;
    const int c = tmp % C;
    const int n = tmp / C;
    const int dstart = dout * stride_d - pad_d;
    const int hstart = hout * stride_h - pad_h;
    const int wstart = wout * stride_w - pad_w;
    int dend = min(dstart + kernel_d, D + pad_d);
    int hend = min(hstart + kernel_h, H + pad_h);
    int wend = min(wstart + kernel_w, W + pad_w);
    const AccT area = (dend - dstart) * (hend - hstart) * (wend - wstart);
    dend = min(dend, D), hend = min(hend, H), wend = min(wend, W);
    const T* offset_x = x + (n * C + c) * D * H * W;
    AccT val = AccT(0);
    for (int d = max(dstart, 0); d < dend; ++d) {
      for (int h = max(hstart, 0); h < hend; ++h) {
        for (int w = max(wstart, 0); w < wend; ++w) {
          val += math::utils::LDGC<AccT>(offset_x + (d * H + h) * W + w);
        }
      }
    }
    y[yi] = val / area;
  }
}

template <typename T, typename AccT>
__global__ void _AvgPool3dNHWC(
    const int nthreads,
    const int C,
    const int D,
    const int H,
    const int W,
    const int out_d,
    const int out_h,
    const int out_w,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_d,
    const int pad_h,
    const int pad_w,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    int tmp = yi / C;
    const int c = yi % C;
    const int wout = tmp % out_w;
    tmp /= out_w;
    const int hout = tmp % out_h;
    tmp /= out_h;
    const int dout = tmp % out_d;
    const int n = tmp / out_d;
    const int dstart = dout * stride_d - pad_d;
    const int hstart = hout * stride_h - pad_h;
    const int wstart = wout * stride_w - pad_w;
    int dend = min(dstart + kernel_d, D + pad_d);
    int hend = min(hstart + kernel_h, H + pad_h);
    int wend = min(wstart + kernel_w, W + pad_w);
    const AccT area = (dend - dstart) * (hend - hstart) * (wend - wstart);
    dend = min(dend, D), hend = min(hend, H), wend = min(wend, W);
    const T* offset_x = x + n * D * H * W * C + c;
    AccT val = AccT(0);
    for (int d = max(dstart, 0); d < dend; ++d) {
      for (int h = max(hstart, 0); h < hend; ++h) {
        for (int w = max(wstart, 0); w < wend; ++w) {
          val += math::utils::LDGC<AccT>(offset_x + ((d * H + h) * W + w) * C);
        }
      }
    }
    y[yi] = val / area;
  }
}

template <typename T, typename AccT>
__global__ void _AvgPool3dGradNCHW(
    const int nthreads,
    const int C,
    const int D,
    const int H,
    const int W,
    const int out_d,
    const int out_h,
    const int out_w,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_d,
    const int pad_h,
    const int pad_w,
    const T* dy,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(xi, nthreads) {
    int tmp = xi / W;
    const int w = xi % W + pad_w;
    const int h = tmp % H + pad_h;
    tmp /= H;
    const int d = tmp % D + pad_d;
    tmp /= D;
    const int c = tmp % C;
    const int n = tmp / C;
    const int out_dstart = d < kernel_d ? 0 : (d - kernel_d) / stride_d + 1;
    const int out_hstart = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
    const int out_wstart = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
    const int out_dend = min(d / stride_d + 1, out_d);
    const int out_hend = min(h / stride_h + 1, out_h);
    const int out_wend = min(w / stride_w + 1, out_w);
    const T* offset_dy = dy + (n * C + c) * out_d * out_h * out_w;
    AccT val = AccT(0);
    for (int dout = out_dstart; dout < out_dend; ++dout) {
      const int dstart = dout * stride_d - pad_d;
      const int dend = min(dstart + kernel_d, D + pad_d);
      for (int hout = out_hstart; hout < out_hend; ++hout) {
        const int hstart = hout * stride_h - pad_h;
        const int hend = min(hstart + kernel_h, H + pad_h);
        for (int wout = out_wstart; wout < out_wend; ++wout) {
          const int wstart = wout * stride_w - pad_w; // clang-format off
          const int wend = min(wstart + kernel_w, W + pad_w);
          const AccT area = (dend - dstart) * (hend - hstart) * (wend - wstart);
          val += math::utils::LDGC<AccT>(offset_dy + (dout * out_h + hout) * out_w + wout) / area;
        } // clang-format on
      }
    }
    dx[xi] = val;
  }
}

template <typename T, typename AccT>
__global__ void _AvgPool3dGradNHWC(
    const int nthreads,
    const int C,
    const int D,
    const int H,
    const int W,
    const int out_d,
    const int out_h,
    const int out_w,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_d,
    const int pad_h,
    const int pad_w,
    const T* dy,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(xi, nthreads) {
    int tmp = xi / C;
    const int c = xi % C;
    const int w = tmp % W + pad_w;
    tmp /= W;
    const int h = tmp % H + pad_h;
    tmp /= H;
    const int d = tmp % D + pad_d;
    const int n = tmp / D;
    const int out_dstart = d < kernel_d ? 0 : (d - kernel_d) / stride_d + 1;
    const int out_hstart = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
    const int out_wstart = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
    const int out_dend = min(d / stride_d + 1, out_d);
    const int out_hend = min(h / stride_h + 1, out_h);
    const int out_wend = min(w / stride_w + 1, out_w);
    const T* offset_dy = dy + n * out_d * out_h * out_w * C + c;
    AccT val = AccT(0);
    for (int dout = out_dstart; dout < out_dend; ++dout) {
      const int dstart = dout * stride_d - pad_d;
      const int dend = min(dstart + kernel_d, D + pad_d);
      for (int hout = out_hstart; hout < out_hend; ++hout) {
        const int hstart = hout * stride_h - pad_h;
        const int hend = min(hstart + kernel_h, H + pad_h);
        for (int wout = out_wstart; wout < out_wend; ++wout) {
          const int wstart = wout * stride_w - pad_w; // clang-format off
          const int wend = min(wstart + kernel_w, W + pad_w);
          const AccT area = (dend - dstart) * (hend - hstart) * (wend - wstart);
          val += math::utils::LDGC<AccT>(offset_dy + ((dout * out_h + hout) * out_w + wout) * C) / area;
        } // clang-format on
      }
    }
    dx[xi] = val;
  }
}

} // namespace

#define DISPATCH_POOL_KERNEL(name, T, AccT, kBlocks, kThreads, ...)  \
  if (data_format == "NCHW") {                                       \
    name##NCHW<T, AccT>                                              \
        <<<kBlocks, kThreads, 0, ctx->hip_stream()>>>(__VA_ARGS__); \
  } else if (data_format == "NHWC") {                                \
    name##NHWC<T, AccT>                                              \
        <<<kBlocks, kThreads, 0, ctx->hip_stream()>>>(__VA_ARGS__); \
  } else {                                                           \
    LOG(FATAL) << "Unknown DataFormat: " << data_format;             \
  }

#define DEFINE_KERNEL_LAUNCHER(name, T, out_dim)                  \
  template <>                                                     \
  void name<T, HIPContext>(                                      \
      const int N,                                                \
      const int C,                                                \
      const int H,                                                \
      const int W,                                                \
      const int out_h,                                            \
      const int out_w,                                            \
      const int kernel_h,                                         \
      const int kernel_w,                                         \
      const int stride_h,                                         \
      const int stride_w,                                         \
      const int pad_h,                                            \
      const int pad_w,                                            \
      const string& data_format,                                  \
      const T* x,                                                 \
      T* y,                                                       \
      HIPContext* ctx) {                                         \
    const int nthreads = N * C * out_dim;                         \
    DISPATCH_POOL_KERNEL(                                         \
        _##name,                                                  \
        math::Traits<T>::scalar_type,                             \
        math::Traits<T>::accumulator_type,                        \
        CUDA_BLOCKS(nthreads),                                    \
        CUDA_THREADS,                                             \
        nthreads,                                                 \
        C,                                                        \
        H,                                                        \
        W,                                                        \
        out_h,                                                    \
        out_w,                                                    \
        kernel_h,                                                 \
        kernel_w,                                                 \
        stride_h,                                                 \
        stride_w,                                                 \
        pad_h,                                                    \
        pad_w,                                                    \
        reinterpret_cast<const math::Traits<T>::scalar_type*>(x), \
        reinterpret_cast<math::Traits<T>::scalar_type*>(y));      \
  }

DEFINE_KERNEL_LAUNCHER(AvgPool2d, float16, (out_h * out_w));
DEFINE_KERNEL_LAUNCHER(AvgPool2d, bfloat16, (out_h * out_w));
DEFINE_KERNEL_LAUNCHER(AvgPool2d, float, (out_h * out_w));
DEFINE_KERNEL_LAUNCHER(AvgPool2d, double, (out_h * out_w));
DEFINE_KERNEL_LAUNCHER(AvgPool2dGrad, float16, (H * W));
DEFINE_KERNEL_LAUNCHER(AvgPool2dGrad, bfloat16, (H * W));
DEFINE_KERNEL_LAUNCHER(AvgPool2dGrad, float, (H * W));
DEFINE_KERNEL_LAUNCHER(AvgPool2dGrad, double, (H * W));
#undef DEFINE_KERNEL_LAUNCHER

#define DEFINE_KERNEL_LAUNCHER(name, T, out_dim)                  \
  template <>                                                     \
  void name<T, HIPContext>(                                      \
      const int N,                                                \
      const int C,                                                \
      const int D,                                                \
      const int H,                                                \
      const int W,                                                \
      const int out_d,                                            \
      const int out_h,                                            \
      const int out_w,                                            \
      const int kernel_d,                                         \
      const int kernel_h,                                         \
      const int kernel_w,                                         \
      const int stride_d,                                         \
      const int stride_h,                                         \
      const int stride_w,                                         \
      const int pad_d,                                            \
      const int pad_h,                                            \
      const int pad_w,                                            \
      const string& data_format,                                  \
      const T* x,                                                 \
      T* y,                                                       \
      HIPContext* ctx) {                                         \
    const int nthreads = N * C * out_dim;                         \
    DISPATCH_POOL_KERNEL(                                         \
        _##name,                                                  \
        math::Traits<T>::scalar_type,                             \
        math::Traits<T>::accumulator_type,                        \
        CUDA_BLOCKS(nthreads),                                    \
        CUDA_THREADS,                                             \
        nthreads,                                                 \
        C,                                                        \
        D,                                                        \
        H,                                                        \
        W,                                                        \
        out_d,                                                    \
        out_h,                                                    \
        out_w,                                                    \
        kernel_d,                                                 \
        kernel_h,                                                 \
        kernel_w,                                                 \
        stride_d,                                                 \
        stride_h,                                                 \
        stride_w,                                                 \
        pad_d,                                                    \
        pad_h,                                                    \
        pad_w,                                                    \
        reinterpret_cast<const math::Traits<T>::scalar_type*>(x), \
        reinterpret_cast<math::Traits<T>::scalar_type*>(y));      \
  }

DEFINE_KERNEL_LAUNCHER(AvgPool3d, float16, (out_d * out_h * out_w));
DEFINE_KERNEL_LAUNCHER(AvgPool3d, bfloat16, (out_d * out_h * out_w));
DEFINE_KERNEL_LAUNCHER(AvgPool3d, float, (out_d * out_h * out_w));
DEFINE_KERNEL_LAUNCHER(AvgPool3d, double, (out_d * out_h * out_w));
DEFINE_KERNEL_LAUNCHER(AvgPool3dGrad, float16, (D * H * W));
DEFINE_KERNEL_LAUNCHER(AvgPool3dGrad, bfloat16, (D * H * W));
DEFINE_KERNEL_LAUNCHER(AvgPool3dGrad, float, (D * H * W));
DEFINE_KERNEL_LAUNCHER(AvgPool3dGrad, double, (D * H * W));
#undef DEFINE_KERNEL_LAUNCHER
#undef DISPATCH_POOL_KERNEL

} // namespace kernels

} // namespace dragon
