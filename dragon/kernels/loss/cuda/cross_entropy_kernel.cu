#include "hip/hip_runtime.h"
#include "dragon/kernels/loss/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T>
__global__ void
_CrossEntropy(const int N, const T* input, const T* target, T* loss) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    loss[i] = -target[i] * log(max(input[i], FLT_MIN));
  }
}

template <typename InputT, typename TargetT>
__global__ void _CrossEntropy(
    const int NxS,
    const int S,
    const int C,
    const int ignore_index,
    const InputT* input,
    const TargetT* target,
    InputT* loss,
    InputT* mask) {
  CUDA_1D_KERNEL_LOOP(index, NxS) {
    const int i = index / S, j = index % S;
    const int tgt = target[index];
    if (tgt == ignore_index) {
      loss[index] = mask[index] = InputT(0);
    } else {
      loss[index] = -log(max(input[(i * C + tgt) * S + j], InputT(FLT_MIN)));
      mask[index] = InputT(1);
    }
  }
}

template <typename T>
__global__ void _SigmoidCrossEntropy(
    const int N,
    const T* input,
    const T* target,
    T* loss,
    T* mask) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    if (target[i] < 0) {
      loss[i] = mask[i] = T(0);
    } else {
      const float lgt = input[i];
      loss[i] = log(1.f + exp(lgt - 2.f * lgt * (lgt >= 0.f))) +
          lgt * ((lgt >= 0.f) - float(target[i]));
      mask[i] = T(1);
    }
  }
}

template <typename T>
__global__ void _SigmoidCrossEntropyGrad(
    const int N,
    const T* input,
    const T* target,
    T* dx,
    T* mask) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    if (target[i] < 0) {
      dx[i] = mask[i] = T(0);
    } else {
      dx[i] = T(1) / (T(1) + exp(-input[i])) - target[i];
      mask[i] = T(1);
    }
  }
}

template <typename InputT, typename TargetT>
__global__ void _SoftmaxCrossEntropyGrad(
    const int NxS,
    const int S,
    const int C,
    const int ignore_index,
    const InputT* /* input */,
    const TargetT* target,
    InputT* dx,
    InputT* mask) {
  CUDA_1D_KERNEL_LOOP(index, NxS) {
    const int i = index / S, j = index % S;
    const int tgt = target[index];
    if (tgt == ignore_index) {
      InputT* offset_dx = dx + i * C * S + j;
      for (int _ = 0; _ < C; ++_, offset_dx += S) {
        offset_dx[0] = InputT(0);
      }
      mask[index] = InputT(0);
    } else {
      dx[(i * C + tgt) * S + j] -= InputT(1);
      mask[index] = InputT(1);
    }
  }
}

} // namespace

#define DEFINE_KERNEL_LAUNCHER(name, T)                               \
  template <>                                                         \
  void name<T, HIPContext>(                                          \
      const int N,                                                    \
      const T* input,                                                 \
      const T* target,                                                \
      T* loss,                                                        \
      HIPContext* ctx) {                                             \
    _##name<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        N, input, target, loss);                                      \
  }

DEFINE_KERNEL_LAUNCHER(CrossEntropy, float);
DEFINE_KERNEL_LAUNCHER(CrossEntropy, double);
#undef DEFINE_KERNEL_LAUNCHER

#define DEFINE_KERNEL_LAUNCHER(name, T)                               \
  template <>                                                         \
  void name<T, HIPContext>(                                          \
      const int N,                                                    \
      const T* input,                                                 \
      const T* target,                                                \
      T* loss,                                                        \
      T* mask,                                                        \
      HIPContext* ctx) {                                             \
    _##name<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        N, input, target, loss, mask);                                \
  }

DEFINE_KERNEL_LAUNCHER(SigmoidCrossEntropy, float);
DEFINE_KERNEL_LAUNCHER(SigmoidCrossEntropy, double);
DEFINE_KERNEL_LAUNCHER(SigmoidCrossEntropyGrad, float);
DEFINE_KERNEL_LAUNCHER(SigmoidCrossEntropyGrad, double);
#undef DEFINE_KERNEL_LAUNCHER

#define DEFINE_KERNEL_LAUNCHER(name, InputT, TargetT)                   \
  template <>                                                           \
  void name<InputT, TargetT, HIPContext>(                              \
      const int N,                                                      \
      const int S,                                                      \
      const int C,                                                      \
      const int ignore_index,                                           \
      const InputT* input,                                              \
      const TargetT* target,                                            \
      InputT* loss,                                                     \
      InputT* mask,                                                     \
      HIPContext* ctx) {                                               \
    const auto NxS = N * S;                                             \
    _##name<<<CUDA_BLOCKS(NxS), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        NxS, S, C, ignore_index, input, target, loss, mask);            \
  }

DEFINE_KERNEL_LAUNCHER(CrossEntropy, float, int);
DEFINE_KERNEL_LAUNCHER(CrossEntropy, float, int64_t);
DEFINE_KERNEL_LAUNCHER(CrossEntropy, double, int);
DEFINE_KERNEL_LAUNCHER(CrossEntropy, double, int64_t);
DEFINE_KERNEL_LAUNCHER(SoftmaxCrossEntropyGrad, float, int);
DEFINE_KERNEL_LAUNCHER(SoftmaxCrossEntropyGrad, float, int64_t);
DEFINE_KERNEL_LAUNCHER(SoftmaxCrossEntropyGrad, double, int);
DEFINE_KERNEL_LAUNCHER(SoftmaxCrossEntropyGrad, double, int64_t);
#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon
