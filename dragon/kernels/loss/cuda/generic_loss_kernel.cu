#include "hip/hip_runtime.h"
#include "dragon/kernels/loss/op_kernels.h"
#include "dragon/utils/math_functions.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, typename AccT>
__global__ void
_ReduceLossGrad(const int N, const AccT scale, const T* dl, T* dx) {
  const AccT alpha = math::utils::LDGC<AccT>(dl) * scale;
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = convert::To<AccT>(dx[i]) * alpha;
  }
}

template <typename T, typename AccT>
__global__ void
_ReduceLossGrad(const int N, const T* inv_scale, const T* dl, T* dx) {
  const AccT alpha = math::utils::LDGC<AccT>(dl) /
      max(math::utils::LDGC<AccT>(inv_scale), AccT(0.5));
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = convert::To<AccT>(dx[i]) * alpha;
  }
}

template <typename T, typename AccT>
__global__ void _BroadcastLossGrad(
    const int NxCxS,
    const int CxS,
    const int S,
    const T* dl,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(i, NxCxS) {
    const AccT val = math::utils::LDGC<AccT>(dl + i / CxS * S + i % S);
    dx[i] = convert::To<AccT>(dx[i]) * val;
  }
}

} // namespace

#define DEFINE_KERNEL_LAUNCHER(T)                               \
  template <>                                                   \
  void ReduceLoss<T, HIPContext>(                              \
      const int N,                                              \
      const int num_masks,                                      \
      const float normalizer,                                   \
      const T* x,                                               \
      const T* mask,                                            \
      T* y,                                                     \
      HIPContext* ctx) {                                       \
    if (num_masks > 0 && normalizer < 0.f) {                    \
      auto* num_valid = const_cast<T*>(mask + num_masks);       \
      math::Sum(num_masks, 1.f, mask, num_valid, ctx);          \
      math::Sum(N, 1.f, x, y, ctx);                             \
      math::Div(1, y, num_valid, y, ctx);                       \
    } else {                                                    \
      math::Sum(N, 1.f / std::max(1.f, normalizer), x, y, ctx); \
    }                                                           \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T)                                         \
  template <>                                                                  \
  void ReduceLossGrad<T, HIPContext>(                                         \
      const int N,                                                             \
      const int num_masks,                                                     \
      const float normalizer,                                                  \
      const T* dl,                                                             \
      const T* mask,                                                           \
      T* dx,                                                                   \
      HIPContext* ctx) {                                                      \
    using ScalarT = math::Traits<T>::scalar_type;                              \
    using AccT = math::Traits<T>::accumulator_type;                            \
    if (num_masks > 0 && normalizer < 0.f) {                                   \
      auto* num_valid = const_cast<T*>(mask + num_masks);                      \
      math::Sum(num_masks, 1.f, mask, num_valid, ctx);                         \
      _ReduceLossGrad<ScalarT, AccT>                                           \
          <<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(           \
              N, (const ScalarT*)num_valid, (const ScalarT*)dl, (ScalarT*)dx); \
    } else {                                                                   \
      const auto scale = AccT(1.f / std::max(0.5f, normalizer));               \
      _ReduceLossGrad<ScalarT, AccT>                                           \
          <<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(           \
              N, scale, (const ScalarT*)dl, (ScalarT*)dx);                     \
    }                                                                          \
  }                                                                            \
  template <>                                                                  \
  void BroadcastLossGrad<T, HIPContext>(                                      \
      const int N,                                                             \
      const int S,                                                             \
      const int C,                                                             \
      const T* dl,                                                             \
      T* dx,                                                                   \
      HIPContext* ctx) {                                                      \
    using ScalarT = math::Traits<T>::scalar_type;                              \
    using AccT = math::Traits<T>::accumulator_type;                            \
    const auto CxS = C * S;                                                    \
    const auto NxCxS = N * CxS;                                                \
    _BroadcastLossGrad<ScalarT, AccT>                                          \
        <<<CUDA_BLOCKS(NxCxS), CUDA_THREADS, 0, ctx->hip_stream()>>>(         \
            NxCxS, CxS, S, (const ScalarT*)dl, (ScalarT*)dx);                  \
  }

DEFINE_KERNEL_LAUNCHER(float16);
DEFINE_KERNEL_LAUNCHER(bfloat16);
DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);
DEFINE_GRAD_KERNEL_LAUNCHER(float16);
DEFINE_GRAD_KERNEL_LAUNCHER(bfloat16);
DEFINE_GRAD_KERNEL_LAUNCHER(float);
DEFINE_GRAD_KERNEL_LAUNCHER(double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon
