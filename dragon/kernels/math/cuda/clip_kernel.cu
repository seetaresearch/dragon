#include "hip/hip_runtime.h"
#include "dragon/kernels/math/op_kernels.h"
#include "dragon/utils/conversions.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, typename AccT>
__global__ void
_Clip(const int N, const AccT low, const AccT high, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = convert::To<T>(max(low, min(convert::To<AccT>(x[i]), high)));
  }
}

template <typename T, typename AccT>
__global__ void _ClipGrad(
    const int N,
    const AccT low,
    const AccT high,
    const T* dy,
    const T* x,
    T* dx) {
  const T kZero = convert::To<T>(0.f);
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT val = convert::To<AccT>(x[i]);
    dx[i] = val < low || val > high ? kZero : dy[i];
  }
}

} // namespace

#define DEFINE_KERNEL_LAUNCHER(T, AccT)                             \
  template <>                                                       \
  void Clip<T, HIPContext>(                                        \
      const int N,                                                  \
      const float low,                                              \
      const float high,                                             \
      const T* x,                                                   \
      T* y,                                                         \
      HIPContext* ctx) {                                           \
    _Clip<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        N, low, high, x, y);                                        \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T, AccT)                            \
  template <>                                                           \
  void ClipGrad<T, HIPContext>(                                        \
      const int N,                                                      \
      const float low,                                                  \
      const float high,                                                 \
      const T* dy,                                                      \
      const T* x,                                                       \
      T* dx,                                                            \
      HIPContext* ctx) {                                               \
    _ClipGrad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        N, low, high, dy, x, dx);                                       \
  }

DEFINE_KERNEL_LAUNCHER(uint8_t, uint8_t);
DEFINE_KERNEL_LAUNCHER(int8_t, int8_t);
DEFINE_KERNEL_LAUNCHER(int, int);
DEFINE_KERNEL_LAUNCHER(int64_t, int64_t);
DEFINE_KERNEL_LAUNCHER(float16, float);
DEFINE_KERNEL_LAUNCHER(float, float);
DEFINE_KERNEL_LAUNCHER(double, double);
DEFINE_GRAD_KERNEL_LAUNCHER(float16, float);
DEFINE_GRAD_KERNEL_LAUNCHER(float, float);
DEFINE_GRAD_KERNEL_LAUNCHER(double, double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon
