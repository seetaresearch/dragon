#include "hip/hip_runtime.h"
#include "dragon/kernels/math/op_kernels.h"
#include "dragon/utils/math_functions.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T>
__global__ void _CosGrad(const int N, const T* dy, const T* x, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = -dy[i] * sin(x[i]);
  }
}

template <>
__global__ void
_CosGrad<half>(const int N, const half* dy, const half* x, half* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = __float2half(-__half2float(dy[i]) * sin(__half2float(x[i])));
  }
} // CosGrad

template <>
__global__ void
_CosGrad<half2>(const int N, const half2* dy, const half2* x, half2* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float2 val = __half22float2(x[i]);
    const float2 grad = __half22float2(dy[i]);
    dx[i] = __floats2half2_rn(-grad.x * sin(val.x), -grad.y * sin(val.y));
  }
} // CosGrad

template <typename T>
__global__ void _SinGrad(const int N, const T* dy, const T* x, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = dy[i] * cos(x[i]);
  }
}

template <>
__global__ void
_SinGrad<half>(const int N, const half* dy, const half* x, half* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = __float2half(__half2float(dy[i]) * cos(__half2float(x[i])));
  }
} // SinGrad

template <>
__global__ void
_SinGrad<half2>(const int N, const half2* dy, const half2* x, half2* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float2 val = __half22float2(x[i]);
    const float2 grad = __half22float2(dy[i]);
    dx[i] = __floats2half2_rn(grad.x * cos(val.x), grad.y * cos(val.y));
  }
} // SinGrad

template <typename T>
__global__ void _ReciprocalGrad(const int N, const T* dy, const T* y, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = -dy[i] * math::utils::Square(y[i]);
  }
}

template <>
__global__ void
_ReciprocalGrad<half>(const int N, const half* dy, const half* y, half* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = __float2half(
        -__half2float(dy[i]) * math::utils::Square(__half2float(y[i])));
  }
} // ReciprocalGrad

template <>
__global__ void _ReciprocalGrad<half2>(
    const int N,
    const half2* dy,
    const half2* y,
    half2* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float2 val = __half22float2(y[i]);
    const float2 grad = __half22float2(dy[i]);
    dx[i] =
        __floats2half2_rn(-grad.x * (val.x * val.x), -grad.y * (val.y * val.y));
  }
} // ReciprocalGrad

template <typename T>
__global__ void _RsqrtGrad(const int N, const T* dy, const T* y, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = T(-0.5) * dy[i] * math::utils::Cube(y[i]);
  }
}

template <>
__global__ void
_RsqrtGrad<half>(const int N, const half* dy, const half* y, half* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = __float2half(
        -0.5f * __half2float(dy[i]) * math::utils::Cube(__half2float(y[i])));
  }
} // ReciprocalGrad

template <>
__global__ void
_RsqrtGrad<half2>(const int N, const half2* dy, const half2* y, half2* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float2 val = __half22float2(y[i]);
    const float2 grad = __half22float2(dy[i]);
    dx[i] = __floats2half2_rn(
        -0.5f * grad.x * (val.x * val.x * val.x),
        -0.5f * grad.y * (val.y * val.y * val.y));
  }
} // ReciprocalGrad

} // namespace

#define DEFINE_GRAD_KERNEL_LAUNCHER(name, T)                                \
  template <>                                                               \
  void name##Grad<T, HIPContext>(                                          \
      const int N, const T* dy, const T* x, T* dx, HIPContext* ctx) {      \
    _##name##Grad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        N, dy, x, dx);                                                      \
  }

DEFINE_GRAD_KERNEL_LAUNCHER(Cos, float);
DEFINE_GRAD_KERNEL_LAUNCHER(Cos, double);
DEFINE_GRAD_KERNEL_LAUNCHER(Sin, float);
DEFINE_GRAD_KERNEL_LAUNCHER(Sin, double);
DEFINE_GRAD_KERNEL_LAUNCHER(Reciprocal, float);
DEFINE_GRAD_KERNEL_LAUNCHER(Reciprocal, double);
DEFINE_GRAD_KERNEL_LAUNCHER(Rsqrt, float);
DEFINE_GRAD_KERNEL_LAUNCHER(Rsqrt, double);
#undef DEFINE_GRAD_KERNEL_LAUNCHER

#define DEFINE_GRAD_KERNEL_LAUNCHER(name)                                     \
  template <>                                                                 \
  void name##Grad<float16, HIPContext>(                                      \
      const int N,                                                            \
      const float16* dy,                                                      \
      const float16* x,                                                       \
      float16* dx,                                                            \
      HIPContext* ctx) {                                                     \
    if ((N & 1) == 0) {                                                       \
      _##name##Grad<<<                                                        \
          CUDA_BLOCKS(N >> 1),                                                \
          CUDA_THREADS,                                                       \
          0,                                                                  \
          ctx->hip_stream()>>>(                                              \
          N >> 1,                                                             \
          reinterpret_cast<const half2*>(dy),                                 \
          reinterpret_cast<const half2*>(x),                                  \
          reinterpret_cast<half2*>(dx));                                      \
    } else {                                                                  \
      _##name##Grad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
          N,                                                                  \
          reinterpret_cast<const half*>(dy),                                  \
          reinterpret_cast<const half*>(x),                                   \
          reinterpret_cast<half*>(dx));                                       \
    }                                                                         \
  }

DEFINE_GRAD_KERNEL_LAUNCHER(Cos);
DEFINE_GRAD_KERNEL_LAUNCHER(Sin);
DEFINE_GRAD_KERNEL_LAUNCHER(Reciprocal);
DEFINE_GRAD_KERNEL_LAUNCHER(Rsqrt);
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon
