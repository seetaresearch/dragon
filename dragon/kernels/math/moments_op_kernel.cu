
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/device/common_cub.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, typename AccT>
__global__ void _RowwiseMoments(
    const int rows,
    const int cols,
    const T* x,
    AccT* mean,
    AccT* var) {
  __shared__ typename BlockReduce<AccT>::TempStorage m_storage;
  __shared__ typename BlockReduce<AccT>::TempStorage v_storage;
  const AccT scale = AccT(1) / AccT(rows);
  CUDA_2D_KERNEL_LOOP1(i, cols) {
    AccT m_val = AccT(0), v_val = AccT(0);
    CUDA_2D_KERNEL_LOOP2(j, rows) {
      const AccT val = convert::To<AccT>(x[j * cols + i]);
      m_val += val;
      v_val += val * val;
    }
    m_val = BlockReduce<AccT>(m_storage).Sum(m_val);
    v_val = BlockReduce<AccT>(v_storage).Sum(v_val);
    if (threadIdx.x == 0) {
      m_val = m_val * scale;
      if (mean != nullptr) mean[i] = m_val;
      var[i] = v_val * scale - m_val * m_val;
    }
  }
}

template <typename T, typename AccT>
__global__ void _ColwiseMoments(
    const int rows,
    const int cols,
    const T* x,
    AccT* mean,
    AccT* var) {
  __shared__ typename BlockReduce<AccT>::TempStorage m_storage;
  __shared__ typename BlockReduce<AccT>::TempStorage v_storage;
  const AccT scale = AccT(1) / AccT(cols);
  CUDA_2D_KERNEL_LOOP1(i, rows) {
    AccT m_val = AccT(0), v_val = AccT(0);
    CUDA_2D_KERNEL_LOOP2(j, cols) {
      const AccT val = convert::To<AccT>(x[i * cols + j]);
      m_val += val;
      v_val += val * val;
    }
    m_val = BlockReduce<AccT>(m_storage).Sum(m_val);
    v_val = BlockReduce<AccT>(v_storage).Sum(v_val);
    if (threadIdx.x == 0) {
      m_val = m_val * scale;
      if (mean != nullptr) mean[i] = m_val;
      var[i] = v_val * scale - m_val * m_val;
    }
  }
}

template <typename T, typename AccT, int D>
__global__ void _GenericMoments(
    const int rows,
    const int cols,
    const SimpleArray<int, D> X_dims,
    const SimpleArray<int, D> X_strides,
    const T* x,
    AccT* mean,
    AccT* var) {
  __shared__ typename BlockReduce<AccT>::TempStorage m_storage;
  __shared__ typename BlockReduce<AccT>::TempStorage v_storage;
  const AccT scale = AccT(1) / AccT(cols);
  CUDA_2D_KERNEL_LOOP1(i, rows) {
    AccT m_val = AccT(0), v_val = AccT(0);
    CUDA_2D_KERNEL_LOOP2(j, cols) {
      int xi = 0, c = i * cols + j;
#pragma unroll
      for (int d = D - 1; d >= 0; --d) {
        int r;
        FIXED_DIVISOR_DIV_MOD(X_dims.data[d], c, &c, &r);
        xi += r * X_strides.data[d];
      }
      const AccT val = convert::To<AccT>(x[xi]);
      m_val += val;
      v_val += val * val;
    }
    m_val = BlockReduce<AccT>(m_storage).Sum(m_val);
    v_val = BlockReduce<AccT>(v_storage).Sum(v_val);
    if (threadIdx.x == 0) {
      mean[i] = m_val = m_val * scale;
      if (mean != nullptr) mean[i] = m_val;
      var[i] = v_val * scale - m_val * m_val;
    }
  }
}

template <typename T, typename AccT, int D>
void _GenericMomentsImpl(
    const int* dims,
    const int num_axes,
    const int* axes,
    const T* x,
    AccT* mean,
    AccT* var,
    CUDAContext* ctx) {
  SimpleArray<int, D> transpose_axes;
  SimpleArray<int, D> transpose_strides;
  SimpleArray<int, D> transpose_dims;
  math::utils::TransposeAxesForReduce(D, num_axes, axes, transpose_axes.data);
  math::utils::ComputeTransposeStrides(
      D, dims, transpose_axes.data, transpose_strides.data);
  int rows = 1, cols = 1;
  const int pivot = D - num_axes;
  for (int i = 0; i < pivot; ++i) {
    rows *= dims[transpose_axes.data[i]];
  }
  for (int i = pivot; i < D; ++i) {
    cols *= dims[transpose_axes.data[i]];
  }
  for (int i = 0; i < D; ++i) {
    transpose_dims.data[i] = dims[transpose_axes.data[i]];
  }
  _GenericMoments<<<rows, CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      rows, cols, transpose_dims, transpose_strides, x, mean, var);
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(T, AccT)                                      \
  template <>                                                                \
  void Moments<T, AccT, CUDAContext>(                                        \
      const int num_dims,                                                    \
      const int64_t* dims,                                                   \
      const int num_axes,                                                    \
      const int64_t* axes,                                                   \
      const T* x,                                                            \
      AccT* mean,                                                            \
      AccT* var,                                                             \
      CUDAContext* ctx) {                                                    \
    vec64_t new_dims, new_axes;                                              \
    math::utils::CollapseReduceAxes(                                         \
        num_dims, dims, num_axes, axes, new_dims, new_axes);                 \
    int num_new_dims = new_dims.size();                                      \
    int64_t rows, cols;                                                      \
    vec64_t out_dims(new_dims);                                              \
    for (auto axis : new_axes) {                                             \
      out_dims[axis] = 1;                                                    \
    }                                                                        \
    if (math::utils::IsRowwiseReduce(                                        \
            num_new_dims, new_dims.data(), out_dims.data(), &rows, &cols)) { \
      return _RowwiseMoments<<<cols, CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
          rows, cols, x, mean, var);                                         \
    }                                                                        \
    if (math::utils::IsColwiseReduce(                                        \
            num_new_dims, new_dims.data(), out_dims.data(), &rows, &cols)) { \
      return _ColwiseMoments<<<rows, CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
          rows, cols, x, mean, var);                                         \
    }                                                                        \
    CUDA_TENSOR_DIMS_CHECK(num_new_dims);                                    \
    DISPATCH_FUNC_BY_VALUE_WITH_TYPE_2(                                      \
        _GenericMomentsImpl,                                                 \
        T,                                                                   \
        AccT,                                                                \
        num_new_dims,                                                        \
        vec32_t({new_dims.begin(), new_dims.end()}).data(),                  \
        new_axes.size(),                                                     \
        vec32_t({new_axes.begin(), new_axes.end()}).data(),                  \
        x,                                                                   \
        mean,                                                                \
        var,                                                                 \
        ctx);                                                                \
  }

DEFINE_KERNEL_LAUNCHER(int, float);
DEFINE_KERNEL_LAUNCHER(int64_t, double);
DEFINE_KERNEL_LAUNCHER(float16, float);
DEFINE_KERNEL_LAUNCHER(float, float);
DEFINE_KERNEL_LAUNCHER(double, double);
#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon

#endif // USE_CUDA
