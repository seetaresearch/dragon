
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, typename AccT>
__global__ void _CumSum(
    const int NxS,
    const int S,
    const int C,
    const bool exclusive,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(i, NxS) {
    int offset = i / S * C * S + i % S;
    y[offset] = exclusive ? convert::To<T>(AccT(0)) : x[offset];
    for (int j = 1; j < C; ++j) {
      const int index = offset + S;
      y[index] = convert::To<T>(
          convert::To<AccT>(y[offset]) +
          convert::To<AccT>(x[exclusive ? offset : index]));
      offset = index;
    }
  }
}

template <typename T, typename AccT>
__global__ void _CumSumReverse(
    const int NxS,
    const int S,
    const int C,
    const bool exclusive,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(i, NxS) {
    int offset = (i / S * C + C - 1) * S + i % S;
    y[offset] = exclusive ? convert::To<T>(AccT(0)) : x[offset];
    for (int j = C - 2; j >= 0; --j) {
      const int index = offset - S;
      y[index] = convert::To<T>(
          convert::To<AccT>(y[offset]) +
          convert::To<AccT>(x[exclusive ? offset : index]));
      offset = index;
    }
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(T, AccT)                                \
  template <>                                                          \
  void CumSum<T, CUDAContext>(                                         \
      const int N,                                                     \
      const int S,                                                     \
      const int C,                                                     \
      const bool exclusive,                                            \
      const bool reverse,                                              \
      const T* x,                                                      \
      T* y,                                                            \
      CUDAContext* ctx) {                                              \
    const auto NxS = N * S;                                            \
    if (reverse) {                                                     \
      _CumSumReverse<math::ScalarType<T>::type, AccT>                  \
          <<<CUDA_BLOCKS(NxS), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
              NxS,                                                     \
              S,                                                       \
              C,                                                       \
              exclusive,                                               \
              reinterpret_cast<const math::ScalarType<T>::type*>(x),   \
              reinterpret_cast<math::ScalarType<T>::type*>(y));        \
    } else {                                                           \
      _CumSum<math::ScalarType<T>::type, AccT>                         \
          <<<CUDA_BLOCKS(NxS), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
              NxS,                                                     \
              S,                                                       \
              C,                                                       \
              exclusive,                                               \
              reinterpret_cast<const math::ScalarType<T>::type*>(x),   \
              reinterpret_cast<math::ScalarType<T>::type*>(y));        \
    }                                                                  \
  }

DEFINE_KERNEL_LAUNCHER(uint8_t, uint8_t);
DEFINE_KERNEL_LAUNCHER(int8_t, int8_t);
DEFINE_KERNEL_LAUNCHER(int, int);
DEFINE_KERNEL_LAUNCHER(int64_t, int64_t);
DEFINE_KERNEL_LAUNCHER(float16, float);
DEFINE_KERNEL_LAUNCHER(float, float);
DEFINE_KERNEL_LAUNCHER(double, double);
#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon

#endif // USE_CUDA
