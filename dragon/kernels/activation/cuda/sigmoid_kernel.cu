#include "hip/hip_runtime.h"
#include "dragon/kernels/activation/op_kernels.h"
#include "dragon/utils/math_functions.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T>
__global__ void _Sigmoid(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = T(1) / (T(1) + exp(-x[i]));
  }
}

template <>
__global__ void _Sigmoid<half>(const int N, const half* x, half* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = __float2half(1.f / (1.f + exp(-__half2float(x[i]))));
  }
}

template <>
__global__ void _Sigmoid<half2>(const int N, const half2* x, half2* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float2 val = __half22float2(x[i]);
    y[i] =
        __floats2half2_rn(1.f / (1.f + exp(-val.x)), 1.f / (1.f + exp(-val.y)));
  }
}

template <typename T, typename AccT>
__global__ void
_HardSigmoid(const int N, const AccT alpha, const AccT beta, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT s_val = fma(convert::To<AccT>(x[i]), alpha, beta);
    y[i] = convert::To<T>(max(AccT(0), min(AccT(1), s_val)));
  }
}

template <typename T>
__global__ void _SigmoidGrad(const int N, const T* dy, const T* y, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = dy[i] * __ldg(y + i) * (1 - __ldg(y + i));
  }
}

template <>
__global__ void
_SigmoidGrad<half>(const int N, const half* dy, const half* y, half* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float val = __half2float(y[i]);
    dx[i] = __float2half(__half2float(dy[i]) * val * (1.f - val));
  }
} // SigmoidGrad

template <>
__global__ void
_SigmoidGrad<half2>(const int N, const half2* dy, const half2* y, half2* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float2 val = __half22float2(y[i]);
    const float2 grad = __half22float2(dy[i]);
    dx[i] = __floats2half2_rn(
        grad.x * val.x * (1.f - val.x), grad.y * val.y * (1.f - val.y));
  }
} // SigmoidGrad

template <typename T, typename AccT>
__global__ void _HardSigmoidGrad(
    const int N,
    const AccT alpha,
    const T* dy,
    const T* y,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT val = convert::To<AccT>(y[i]);
    dx[i] = convert::To<T>(
        (val > AccT(0) && val < AccT(1)) ? convert::To<AccT>(dy[i]) * alpha
                                         : AccT(0));
  }
}

} // namespace

template <>
void Sigmoid<float16, HIPContext>(
    const int N,
    const float16* x,
    float16* y,
    HIPContext* ctx) {
  if ((N & 1) == 0) {
    _Sigmoid<<<CUDA_BLOCKS(N >> 1), CUDA_THREADS, 0, ctx->hip_stream()>>>(
        N >> 1, reinterpret_cast<const half2*>(x), reinterpret_cast<half2*>(y));
  } else {
    _Sigmoid<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(
        N, reinterpret_cast<const half*>(x), reinterpret_cast<half*>(y));
  }
}

template <>
void SigmoidGrad<float16, HIPContext>(
    const int N,
    const float16* dy,
    const float16* y,
    float16* dx,
    HIPContext* ctx) {
  if ((N & 1) == 0) {
    _SigmoidGrad<<<CUDA_BLOCKS(N >> 1), CUDA_THREADS, 0, ctx->hip_stream()>>>(
        N >> 1,
        reinterpret_cast<const half2*>(dy),
        reinterpret_cast<const half2*>(y),
        reinterpret_cast<half2*>(dx));
  } else {
    _SigmoidGrad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(
        N,
        reinterpret_cast<const half*>(dy),
        reinterpret_cast<const half*>(y),
        reinterpret_cast<half*>(dx));
  }
} // SigmoidGrad

#define DEFINE_KERNEL_LAUNCHER(T)                                      \
  template <>                                                          \
  void Sigmoid<T, HIPContext>(                                        \
      const int N, const T* x, T* y, HIPContext* ctx) {               \
    _Sigmoid<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        N, x, y);                                                      \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T)                                     \
  template <>                                                              \
  void SigmoidGrad<T, HIPContext>(                                        \
      const int N, const T* dy, const T* y, T* dx, HIPContext* ctx) {     \
    _SigmoidGrad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        N, dy, y, dx);                                                     \
  }

DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);
DEFINE_GRAD_KERNEL_LAUNCHER(float);
DEFINE_GRAD_KERNEL_LAUNCHER(double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

#define DEFINE_KERNEL_LAUNCHER(T)                                          \
  template <>                                                              \
  void HardSigmoid<T, HIPContext>(                                        \
      const int N,                                                         \
      const float alpha,                                                   \
      const float beta,                                                    \
      const T* x,                                                          \
      T* y,                                                                \
      HIPContext* ctx) {                                                  \
    _HardSigmoid<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        N,                                                                 \
        convert::To<math::AccumulatorType<T>::type>(alpha),                \
        convert::To<math::AccumulatorType<T>::type>(beta),                 \
        reinterpret_cast<const math::ScalarType<T>::type*>(x),             \
        reinterpret_cast<math::ScalarType<T>::type*>(y));                  \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T)                                         \
  template <>                                                                  \
  void HardSigmoidGrad<T, HIPContext>(                                        \
      const int N,                                                             \
      const float alpha,                                                       \
      const T* dy,                                                             \
      const T* y,                                                              \
      T* dx,                                                                   \
      HIPContext* ctx) {                                                      \
    _HardSigmoidGrad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        N,                                                                     \
        convert::To<math::AccumulatorType<T>::type>(alpha),                    \
        reinterpret_cast<const math::ScalarType<T>::type*>(dy),                \
        reinterpret_cast<const math::ScalarType<T>::type*>(y),                 \
        reinterpret_cast<math::ScalarType<T>::type*>(dx));                     \
  }

DEFINE_KERNEL_LAUNCHER(float16);
DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);
DEFINE_GRAD_KERNEL_LAUNCHER(float16);
DEFINE_GRAD_KERNEL_LAUNCHER(float);
DEFINE_GRAD_KERNEL_LAUNCHER(double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon
