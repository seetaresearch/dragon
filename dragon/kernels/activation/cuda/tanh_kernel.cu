#include "hip/hip_runtime.h"
#include "dragon/kernels/activation/op_kernels.h"
#include "dragon/utils/math_functions.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T>
__global__ void _Tanh(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = tanh(x[i]);
  }
}

template <>
__global__ void _Tanh<half>(const int N, const half* x, half* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = __float2half(tanh(__half2float(x[i])));
  }
}

template <>
__global__ void _Tanh<half2>(const int N, const half2* x, half2* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float2 val = __half22float2(x[i]);
    y[i] = __floats2half2_rn(tanh(val.x), tanh(val.y));
  }
}

template <typename T>
__global__ void _TanhGrad(const int N, const T* dy, const T* y, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = dy[i] * (T(1) - math::utils::Square(y[i]));
  }
}

template <>
__global__ void
_TanhGrad<half>(const int N, const half* dy, const half* y, half* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = __float2half(
        __half2float(dy[i]) * (1.f - math::utils::Square(__half2float(y[i]))));
  }
}

template <>
__global__ void
_TanhGrad<half2>(const int N, const half2* dy, const half2* y, half2* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float2 val = __half22float2(y[i]);
    const float2 grad = __half22float2(dy[i]);
    dx[i] = __floats2half2_rn(
        grad.x * (1.f - math::utils::Square(val.x)),
        grad.y * (1.f - math::utils::Square(val.y)));
  }
}

} // namespace

template <>
void Tanh<float16, HIPContext>(
    const int N,
    const float16* x,
    float16* y,
    HIPContext* ctx) {
  if ((N & 1) == 0) {
    _Tanh<<<CUDA_BLOCKS(N >> 1), CUDA_THREADS, 0, ctx->hip_stream()>>>(
        N >> 1, reinterpret_cast<const half2*>(x), reinterpret_cast<half2*>(y));
  } else {
    _Tanh<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(
        N, reinterpret_cast<const half*>(x), reinterpret_cast<half*>(y));
  }
}

template <>
void TanhGrad<float16, HIPContext>(
    const int N,
    const float16* dy,
    const float16* y,
    float16* dx,
    HIPContext* ctx) {
  if ((N & 1) == 0) {
    _TanhGrad<<<CUDA_BLOCKS(N >> 1), CUDA_THREADS, 0, ctx->hip_stream()>>>(
        N >> 1,
        reinterpret_cast<const half2*>(dy),
        reinterpret_cast<const half2*>(y),
        reinterpret_cast<half2*>(dx));
  } else {
    _TanhGrad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(
        N,
        reinterpret_cast<const half*>(dy),
        reinterpret_cast<const half*>(y),
        reinterpret_cast<half*>(dx));
  }
} // TanhGrad

#define DEFINE_KERNEL_LAUNCHER(T)                                              \
  template <>                                                                  \
  void Tanh<T, HIPContext>(const int N, const T* x, T* y, HIPContext* ctx) { \
    _Tanh<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(N, x, y);   \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T)                                  \
  template <>                                                           \
  void TanhGrad<T, HIPContext>(                                        \
      const int N, const T* dy, const T* y, T* dx, HIPContext* ctx) {  \
    _TanhGrad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>( \
        N, dy, y, dx);                                                  \
  }

DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);
DEFINE_GRAD_KERNEL_LAUNCHER(float);
DEFINE_GRAD_KERNEL_LAUNCHER(double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon
