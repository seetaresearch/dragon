#include "hip/hip_runtime.h"
#include "dragon/kernels/activation/op_kernels.h"
#include "dragon/utils/math_functions.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, typename AccT>
__global__ void _Gelu(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT val = convert::To<AccT>(x[i]);
    y[i] = convert::To<T>(val * normcdf(val));
  }
}

template <typename T, typename AccT>
__global__ void _GeluGrad(const int N, const T* dy, const T* x, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT val = convert::To<AccT>(x[i]);
    dx[i] = convert::To<T>(
        convert::To<AccT>(dy[i]) *
        fma(AccT(0.398942) * val, exp(val * val * AccT(-0.5)), normcdf(val)));
  }
}

template <typename T, typename AccT>
__global__ void _ApproxGelu(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT val = convert::To<AccT>(x[i]);
    y[i] = fma(val,
               tanh(AccT(0.797885) * fma(AccT(0.044715), val * val * val, val)),
               val) *
        AccT(0.5);
  }
}

template <typename T, typename AccT>
__global__ void _ApproxGeluGrad(const int N, const T* dy, const T* x, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT val = convert::To<AccT>(x[i]);
    const AccT val2 =
        tanh(AccT(0.797885) * fma(AccT(0.044715), val * val * val, val));
    dx[i] = convert::To<T>(
        convert::To<AccT>(dy[i]) * AccT(0.5) *
        fma(fma(-val, val2 * val2, val),
            fma(AccT(0.107032), val * val, AccT(0.797885)),
            val2 + AccT(1)));
  }
}

} // namespace

#define DEFINE_KERNEL_LAUNCHER(name, T)                                        \
  template <>                                                                  \
  void name<T, HIPContext>(const int N, const T* x, T* y, HIPContext* ctx) { \
    _##name<math::ScalarType<T>::type, math::AccumulatorType<T>::type>         \
        <<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(             \
            N,                                                                 \
            reinterpret_cast<const math::ScalarType<T>::type*>(x),             \
            reinterpret_cast<math::ScalarType<T>::type*>(y));                  \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(name, T)                           \
  template <>                                                          \
  void name<T, HIPContext>(                                           \
      const int N, const T* dy, const T* x, T* dx, HIPContext* ctx) { \
    _##name<math::ScalarType<T>::type, math::AccumulatorType<T>::type> \
        <<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(     \
            N,                                                         \
            reinterpret_cast<const math::ScalarType<T>::type*>(dy),    \
            reinterpret_cast<const math::ScalarType<T>::type*>(x),     \
            reinterpret_cast<math::ScalarType<T>::type*>(dx));         \
  }

DEFINE_KERNEL_LAUNCHER(Gelu, float16);
DEFINE_KERNEL_LAUNCHER(Gelu, float);
DEFINE_KERNEL_LAUNCHER(Gelu, double);
DEFINE_KERNEL_LAUNCHER(ApproxGelu, float16);
DEFINE_KERNEL_LAUNCHER(ApproxGelu, float);
DEFINE_KERNEL_LAUNCHER(ApproxGelu, double);
DEFINE_GRAD_KERNEL_LAUNCHER(GeluGrad, float16);
DEFINE_GRAD_KERNEL_LAUNCHER(GeluGrad, float);
DEFINE_GRAD_KERNEL_LAUNCHER(GeluGrad, double);
DEFINE_GRAD_KERNEL_LAUNCHER(ApproxGeluGrad, float16);
DEFINE_GRAD_KERNEL_LAUNCHER(ApproxGeluGrad, float);
DEFINE_GRAD_KERNEL_LAUNCHER(ApproxGeluGrad, double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon
