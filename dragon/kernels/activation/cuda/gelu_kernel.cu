#include "hip/hip_runtime.h"
#include "dragon/kernels/activation/op_kernels.h"
#include "dragon/utils/math_functions.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, typename AccT>
__global__ void _Gelu(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT v = convert::To<AccT>(x[i]);
    y[i] = v * normcdf(v);
  }
}

template <typename T, typename AccT>
__global__ void _ApproxGelu(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT v = convert::To<AccT>(x[i]);
    const AccT s = tanh(AccT(0.797884) * fma(AccT(0.044715), v * v * v, v));
    y[i] = fma(v, s, v) * AccT(0.5);
  }
}

template <typename T, typename AccT>
__global__ void _GeluGrad(const int N, const T* dy, const T* x, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT v = convert::To<AccT>(x[i]);
    dx[i] = convert::To<AccT>(dy[i]) *
        fma(AccT(0.398942) * v, exp(v * v * AccT(-0.5)), normcdf(v));
  }
}

template <typename T, typename AccT>
__global__ void _ApproxGeluGrad(const int N, const T* dy, const T* x, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const AccT v = convert::To<AccT>(x[i]);
    const AccT s = tanh(AccT(0.797884) * fma(AccT(0.044715), v * v * v, v));
    dx[i] = convert::To<AccT>(dy[i]) * AccT(0.5) *
        fma(fma(-v, s * s, v),
            fma(AccT(0.107032), v * v, AccT(0.797884)),
            s + AccT(1));
  }
}

} // namespace

#define DEFINE_KERNEL_LAUNCHER(name, T)                                        \
  template <>                                                                  \
  void name<T, HIPContext>(const int N, const T* x, T* y, HIPContext* ctx) { \
    _##name<math::Traits<T>::scalar_type, math::Traits<T>::accumulator_type>   \
        <<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(             \
            N,                                                                 \
            reinterpret_cast<const math::Traits<T>::scalar_type*>(x),          \
            reinterpret_cast<math::Traits<T>::scalar_type*>(y));               \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(name, T)                                 \
  template <>                                                                \
  void name<T, HIPContext>(                                                 \
      const int N, const T* dy, const T* x, T* dx, HIPContext* ctx) {       \
    _##name<math::Traits<T>::scalar_type, math::Traits<T>::accumulator_type> \
        <<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->hip_stream()>>>(           \
            N,                                                               \
            reinterpret_cast<const math::Traits<T>::scalar_type*>(dy),       \
            reinterpret_cast<const math::Traits<T>::scalar_type*>(x),        \
            reinterpret_cast<math::Traits<T>::scalar_type*>(dx));            \
  }

DEFINE_KERNEL_LAUNCHER(Gelu, float16);
DEFINE_KERNEL_LAUNCHER(Gelu, bfloat16);
DEFINE_KERNEL_LAUNCHER(Gelu, float);
DEFINE_KERNEL_LAUNCHER(Gelu, double);
DEFINE_KERNEL_LAUNCHER(ApproxGelu, float16);
DEFINE_KERNEL_LAUNCHER(ApproxGelu, bfloat16);
DEFINE_KERNEL_LAUNCHER(ApproxGelu, float);
DEFINE_KERNEL_LAUNCHER(ApproxGelu, double);
DEFINE_GRAD_KERNEL_LAUNCHER(GeluGrad, float16);
DEFINE_GRAD_KERNEL_LAUNCHER(GeluGrad, bfloat16);
DEFINE_GRAD_KERNEL_LAUNCHER(GeluGrad, float);
DEFINE_GRAD_KERNEL_LAUNCHER(GeluGrad, double);
DEFINE_GRAD_KERNEL_LAUNCHER(ApproxGeluGrad, float16);
DEFINE_GRAD_KERNEL_LAUNCHER(ApproxGeluGrad, bfloat16);
DEFINE_GRAD_KERNEL_LAUNCHER(ApproxGeluGrad, float);
DEFINE_GRAD_KERNEL_LAUNCHER(ApproxGeluGrad, double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon
