
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T>
__global__ void _RMSprop(
    const int N,
    const T lr,
    const T momentum,
    const T decay,
    const T eps,
    T* g,
    T* m,
    T* v) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    T gi = g[i];
    T vi = v[i] = decay * v[i] + (1 - decay) * gi * gi;
    T mi = m[i] = fma(momentum, m[i], gi / (sqrt(vi) + eps));
    g[i] = lr * mi;
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void RMSprop<float, CUDAContext>(
    const int N,
    const float lr,
    const float momentum,
    const float decay,
    const float eps,
    float* g,
    float* m,
    float* v,
    CUDAContext* ctx) {
  _RMSprop<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      N, lr, momentum, decay, eps, g, m, v);
}

} // namespace kernels

} // namespace dragon

#endif // USE_CUDA
