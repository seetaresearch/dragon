#include "hip/hip_runtime.h"
#include "dragon/kernels/normalization/op_kernels.h"
#include "dragon/utils/device/common_cub.h"
#include "dragon/utils/math_functions.h"

namespace dragon {

namespace kernels {

namespace {

#define LDG(x, i) __ldg(x + i)
#define LDG2(x, i) convert::To<AccT>(__ldg(x + i))

template <typename T, typename AccT, StorageOrder kOrder>
__global__ void _GroupNorm(
    const int NxCxS,
    const int G,
    const int D,
    const int S,
    const T* x,
    const AccT* mu,
    const AccT* rsig,
    const AccT* gamma,
    const AccT* beta,
    T* y) {
  const int C = G * D;
  CUDA_1D_KERNEL_LOOP(i, NxCxS) {
    const int ng = kOrder == StorageOrder::NCHW ? i / (D * S)
                                                : i / (C * S) * G + (i / D % G);
    const int c = kOrder == StorageOrder::NCHW ? i / S % C : i % C;
    y[i] = convert::To<T>(
        fma((convert::To<AccT>(x[i]) - __ldg(mu + ng)) * __ldg(rsig + ng),
            __ldg(gamma + c),
            __ldg(beta + c)));
  }
}

template <typename T, typename AccT, StorageOrder kOrder>
__global__ void _GroupNormWGrad(
    const int N,
    const int G,
    const int D,
    const int S,
    const T* x,
    const AccT* mu,
    const AccT* rsig,
    const T* dy,
    AccT* dgamma,
    AccT* dbeta) {
  const int GxD = G * D;
  const int NxS = N * S;
  __shared__ typename BlockReduce<AccT>::TempStorage dg_storage;
  __shared__ typename BlockReduce<AccT>::TempStorage db_storage;
  CUDA_2D_KERNEL_LOOP1(i, GxD) {
    AccT dg_val = AccT(0), db_val = AccT(0);
    CUDA_2D_KERNEL_LOOP2(j, NxS) {
      const int n = j / S;
      const int ng = n * G + i / D;
      const int idx = kOrder == StorageOrder::NCHW ? (n * GxD + i) * S + j % S
                                                   : j * GxD + i;
      dg_val += LDG2(dy, idx) * (LDG2(x, idx) - LDG(mu, ng)) * LDG(rsig, ng);
      db_val += LDG2(dy, idx);
    }
    dg_val = BlockReduce<AccT>(dg_storage).Sum(dg_val);
    db_val = BlockReduce<AccT>(db_storage).Sum(db_val);
    if (threadIdx.x == 0) {
      dgamma[i] = dg_val;
      dbeta[i] = db_val;
    }
  }
}

template <typename T, typename AccT, StorageOrder kOrder>
__global__ void _GroupNormInternalGrad(
    const int N,
    const int G,
    const int D,
    const int S,
    const T* x,
    const AccT* gamma,
    const T* dy,
    AccT* ds,
    AccT* db) {
  const int NxG = N * G;
  const int DxS = D * S;
  __shared__ typename BlockReduce<AccT>::TempStorage ds_storage;
  __shared__ typename BlockReduce<AccT>::TempStorage db_storage;
  CUDA_2D_KERNEL_LOOP1(i, NxG) {
    AccT ds_val = AccT(0), db_val = AccT(0);
    CUDA_2D_KERNEL_LOOP2(j, DxS) {
      const int c = i % G * D + j / S;
      const int idx = kOrder == StorageOrder::NCHW
          ? i * DxS + j
          : (i / G * S + j % S) * G * D + c;
      ds_val += LDG(gamma, c) * LDG2(dy, idx) * LDG2(x, idx);
      db_val += LDG(gamma, c) * LDG2(dy, idx);
    }
    ds_val = BlockReduce<AccT>(ds_storage).Sum(ds_val);
    db_val = BlockReduce<AccT>(db_storage).Sum(db_val);
    if (threadIdx.x == 0) {
      ds[i] = ds_val;
      db[i] = db_val;
    }
  }
}

template <typename T, typename AccT, StorageOrder kOrder>
__global__ void _GroupNormGrad(
    const int NxCxS,
    const int G,
    const int D,
    const int S,
    const T* x,
    const AccT* mu,
    const AccT* rsig,
    const AccT* gamma,
    const AccT* ds,
    const AccT* db,
    const T* dy,
    T* dx) {
  const int C = G * D;
  const AccT denom = AccT(1) / AccT(D * S);
  CUDA_1D_KERNEL_LOOP(i, NxCxS) {
    const int ng = kOrder == StorageOrder::NCHW ? i / (D * S)
                                                : i / (C * S) * G + (i / D % G);
    const int c = kOrder == StorageOrder::NCHW ? i / S % C : i % C;
    const AccT u = fma(LDG(db, ng), LDG(mu, ng), -LDG(ds, ng)) *
        (LDG2(x, i) - LDG(mu, ng)) * math::utils::Cube(LDG(rsig, ng));
    const AccT v = LDG(db, ng) * LDG(rsig, ng);
    dx[i] = convert::To<T>(
        LDG(gamma, c) * LDG2(dy, i) * LDG(rsig, ng) + (u - v) * denom);
  }
}

#undef LDG
#undef LDG2

} // namespace

#define DISPATCH_GROUPNORM_KERNEL(name, T, AccT, kBlocks, kThreads, ...) \
  if (data_format == "NCHW") {                                           \
    name<T, AccT, StorageOrder::NCHW>                                    \
        <<<kBlocks, kThreads, 0, ctx->hip_stream()>>>(__VA_ARGS__);     \
  } else if (data_format == "NHWC") {                                    \
    name<T, AccT, StorageOrder::NHWC>                                    \
        <<<kBlocks, kThreads, 0, ctx->hip_stream()>>>(__VA_ARGS__);     \
  } else {                                                               \
    LOG(FATAL) << "Unknown DataFormat: " << data_format;                 \
  }

#define DEFINE_KERNEL_LAUNCHER(T, AccT)                        \
  template <>                                                  \
  void GroupNorm<T, AccT, HIPContext>(                        \
      const int N,                                             \
      const int G,                                             \
      const int D,                                             \
      const int S,                                             \
      const string& data_format,                               \
      const T* x,                                              \
      const AccT* mu,                                          \
      const AccT* rsig,                                        \
      const AccT* gamma,                                       \
      const AccT* beta,                                        \
      T* y,                                                    \
      HIPContext* ctx) {                                      \
    const auto NxCxS = N * G * D * S;                          \
    DISPATCH_GROUPNORM_KERNEL(                                 \
        _GroupNorm,                                            \
        math::ScalarType<T>::type,                             \
        AccT,                                                  \
        CUDA_BLOCKS(NxCxS),                                    \
        CUDA_THREADS,                                          \
        NxCxS,                                                 \
        G,                                                     \
        D,                                                     \
        S,                                                     \
        reinterpret_cast<const math::ScalarType<T>::type*>(x), \
        mu,                                                    \
        rsig,                                                  \
        gamma,                                                 \
        beta,                                                  \
        reinterpret_cast<math::ScalarType<T>::type*>(y));      \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T, AccT)                    \
  template <>                                                   \
  void GroupNormGrad<T, AccT, HIPContext>(                     \
      const int N,                                              \
      const int G,                                              \
      const int D,                                              \
      const int S,                                              \
      const string& data_format,                                \
      const T* x,                                               \
      const AccT* mu,                                           \
      const AccT* rsig,                                         \
      const AccT* gamma,                                        \
      const T* dy,                                              \
      AccT* ds,                                                 \
      AccT* db,                                                 \
      AccT* dgamma,                                             \
      AccT* dbeta,                                              \
      T* dx,                                                    \
      HIPContext* ctx) {                                       \
    db = ds == db ? ds + N * G : db;                            \
    const auto NxCxS = N * G * D * S;                           \
    DISPATCH_GROUPNORM_KERNEL(                                  \
        _GroupNormWGrad,                                        \
        math::ScalarType<T>::type,                              \
        AccT,                                                   \
        G* D,                                                   \
        CUDA_THREADS,                                           \
        N,                                                      \
        G,                                                      \
        D,                                                      \
        S,                                                      \
        reinterpret_cast<const math::ScalarType<T>::type*>(x),  \
        mu,                                                     \
        rsig,                                                   \
        reinterpret_cast<const math::ScalarType<T>::type*>(dy), \
        dgamma,                                                 \
        dbeta);                                                 \
    DISPATCH_GROUPNORM_KERNEL(                                  \
        _GroupNormInternalGrad,                                 \
        math::ScalarType<T>::type,                              \
        AccT,                                                   \
        N* G,                                                   \
        CUDA_THREADS,                                           \
        N,                                                      \
        G,                                                      \
        D,                                                      \
        S,                                                      \
        reinterpret_cast<const math::ScalarType<T>::type*>(x),  \
        gamma,                                                  \
        reinterpret_cast<const math::ScalarType<T>::type*>(dy), \
        ds,                                                     \
        db);                                                    \
    DISPATCH_GROUPNORM_KERNEL(                                  \
        _GroupNormGrad,                                         \
        math::ScalarType<T>::type,                              \
        AccT,                                                   \
        CUDA_BLOCKS(NxCxS),                                     \
        CUDA_THREADS,                                           \
        NxCxS,                                                  \
        G,                                                      \
        D,                                                      \
        S,                                                      \
        reinterpret_cast<const math::ScalarType<T>::type*>(x),  \
        mu,                                                     \
        rsig,                                                   \
        gamma,                                                  \
        ds,                                                     \
        db,                                                     \
        reinterpret_cast<const math::ScalarType<T>::type*>(dy), \
        reinterpret_cast<math::ScalarType<T>::type*>(dx));      \
  }

DEFINE_KERNEL_LAUNCHER(float16, float);
DEFINE_KERNEL_LAUNCHER(float, float);
DEFINE_KERNEL_LAUNCHER(double, double);
DEFINE_GRAD_KERNEL_LAUNCHER(float16, float);
DEFINE_GRAD_KERNEL_LAUNCHER(float, float);
DEFINE_GRAD_KERNEL_LAUNCHER(double, double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER
#undef DISPATCH_GROUPNORM_KERNEL

} // namespace kernels

} // namespace dragon
